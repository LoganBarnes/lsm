// SparseMatrixGpu.cpp
#include "lsm/gpu/SparseMatrixGpu.hpp"

#include <iostream>
#include <hip/hip_runtime.h>
#include <thrust/device_vector.h>
#include "gpu/hip/hip_runtime_api.h"


namespace lsm
{


namespace
{

///
/// \brief The CudaEnvironment struct
///
struct CudaEnvironment
{

  CudaEnvironment( )
  {
    if ( findCudaDevice( 0, 0, false ) < 0 )
    {
      throw std::runtime_error( "No CUDA capable devices found" );
    }

#ifdef VERBOSE_PRINT
      std::cout << "CUDA device initialized" << std::endl;
#endif
  }


  ~CudaEnvironment( )
  {
    // hipDeviceReset causes the driver to clean up all state. While
    // not mandatory in normal operation, it is good practice.  It is also
    // needed to ensure correct operation when the application is being
    // profiled. Calling hipDeviceReset causes all profile data to be
    // flushed before the application exits
    hipDeviceReset( );

#ifdef VERBOSE_PRINT
    std::cout << "CUDA device reset" << std::endl;
#endif
  }


};

///
/// \brief cudaEnvironment
///
const CudaEnvironment cudaEnvironment;

} // namespace


template< typename T >
class SparseMatrixGpu< T >::SparseMatrixGpuImpl
{

public:

  explicit
  SparseMatrixGpuImpl();


private:

  thrust::device_vector< T > dataPlaceholder_;

};

template< typename T >
SparseMatrixGpu< T >::SparseMatrixGpuImpl::SparseMatrixGpuImpl()
  : dataPlaceholder_( 10 ) // temp init size
{}


////////////////////////////////////////////////////////
////////////////////////////////////////////////////////

template< typename T >
SparseMatrixGpu< T >::SparseMatrixGpu()
  : upImpl_( new SparseMatrixGpuImpl() )
{}


template< typename T >
SparseMatrixGpu< T >::~SparseMatrixGpu()
{}


} // namespace lsm

template class lsm::SparseMatrixGpu< float >;
template class lsm::SparseMatrixGpu< double >;
